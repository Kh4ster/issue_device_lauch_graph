
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <algorithm>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                              \
    }                                                                          \
}

__global__
void kernel_hat(int row)
{
    printf("Hello from %d\n", row);
}

__global__
void kernel(hipGraphExec_t* graphExecs)
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;

    hipError_t status = hipGraphLaunch(graphExecs[row], cudaStreamGraphFireAndForget);
    if (status != hipSuccess) {
        printf("CUDA API failed for row %d with error: %s (%d)\n", row, hipGetErrorString(status), status);
    }
    else
    {
        printf("Parent call to row %d worked\n", row);
    } 
}

int main()
{
    int blockSize = 128;

    std::vector<hipGraphExec_t> graphExecs(blockSize);
    std::vector<hipGraph_t> graphs(blockSize);
    hipGraphExec_t* d_graphExecs;
    CHECK_CUDA(hipMalloc(&d_graphExecs, blockSize * sizeof(hipGraphExec_t)));
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Init child nodes
    for (int row = 0; row < blockSize; ++row)
    {
        dim3 grid(1);
        dim3 block(1);

        void* void_kernel_args[] = {
            (void*)&row
        };

        CHECK_CUDA(hipGraphCreate(&graphs[row], 0));

        hipKernelNodeParams kernelNodeParams;
        kernelNodeParams.func = (void*)kernel_hat;
        kernelNodeParams.gridDim = grid;
        kernelNodeParams.blockDim = block;
        kernelNodeParams.sharedMemBytes = 0;
        kernelNodeParams.kernelParams = void_kernel_args;
        kernelNodeParams.extra = nullptr;

        hipGraphNode_t kernelNode;
        CHECK_CUDA(hipGraphAddKernelNode(&kernelNode, graphs[row], nullptr, 0, &kernelNodeParams));
    
        CHECK_CUDA(hipGraphInstantiateWithFlags(&graphExecs[row], graphs[row], hipGraphInstantiateFlagDeviceLaunch));
        CHECK_CUDA(hipGraphUpload(graphExecs[row], stream));
    }
    hipMemcpy(d_graphExecs, graphExecs.data(), blockSize * sizeof(hipGraphExec_t), hipMemcpyHostToDevice);
    // --

    // Init parent graph
    hipGraph_t graph;
    hipGraphExec_t instance;
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    kernel<<<1, blockSize, 0, stream>>>(d_graphExecs);
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
    hipGraphUpload(instance, stream);
    // --

    // Launch
    CHECK_CUDA(hipGraphLaunch(instance, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
}